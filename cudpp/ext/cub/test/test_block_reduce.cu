#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2014, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of BlockReduce utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <hip/device_functions.h>

#include <cub/util_allocator.cuh>
#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose       = false;
int                     g_repeat        = 0;
CachingDeviceAllocator  g_allocator(true);



//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------


/// Generic reduction (full, 1)
template <typename BlockReduce, typename T, typename ReductionOp>
__device__ __forceinline__ T DeviceTest(
    typename BlockReduce::TempStorage &temp_storage, T (&data)[1], ReductionOp &reduction_op)
{
    return BlockReduce(temp_storage).Reduce(data[0], reduction_op);
}

/// Generic reduction (full, ITEMS_PER_THREAD)
template <typename BlockReduce, typename T, int ITEMS_PER_THREAD, typename ReductionOp>
__device__ __forceinline__ T DeviceTest(
    typename BlockReduce::TempStorage &temp_storage, T (&data)[ITEMS_PER_THREAD], ReductionOp &reduction_op)
{
    return BlockReduce(temp_storage).Reduce(data, reduction_op);
}

/// Generic reduction (partial, 1)
template <typename BlockReduce, typename T, typename ReductionOp>
__device__ __forceinline__ T DeviceTest(
    typename BlockReduce::TempStorage &temp_storage, T &data, ReductionOp &reduction_op, int valid_threads)
{
    return BlockReduce(temp_storage).Reduce(data, reduction_op, valid_threads);
}

/// Sum reduction (full, 1)
template <typename BlockReduce, typename T>
__device__ __forceinline__ T DeviceTest(
    typename BlockReduce::TempStorage &temp_storage, T (&data)[1], Sum &reduction_op)
{
    return BlockReduce(temp_storage).Sum(data[0]);
}

/// Sum reduction (full, ITEMS_PER_THREAD)
template <typename BlockReduce, typename T, int ITEMS_PER_THREAD>
__device__ __forceinline__ T DeviceTest(
    typename BlockReduce::TempStorage &temp_storage, T (&data)[ITEMS_PER_THREAD], Sum &reduction_op)
{
    return BlockReduce(temp_storage).Sum(data);
}

/// Sum reduction (partial, 1)
template <typename BlockReduce, typename T>
__device__ __forceinline__ T DeviceTest(
    typename BlockReduce::TempStorage &temp_storage, T &data, Sum &reduction_op, int valid_threads)
{
    return BlockReduce(temp_storage).Sum(data, valid_threads);
}


/**
 * Test full-tile reduction kernel (where num_items is an even
 * multiple of BLOCK_THREADS)
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    typename                T,
    typename                ReductionOp>
__launch_bounds__ (BLOCK_THREADS, 1)
__global__ void FullTileReduceKernel(
    T                       *d_in,
    T                       *d_out,
    ReductionOp             reduction_op,
    int                     tiles,
    clock_t                 *d_elapsed)
{
    const int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

    // Cooperative threadblock reduction utility type (returns aggregate in thread 0)
    typedef BlockReduce<T, BLOCK_THREADS, ALGORITHM> BlockReduce;

    // Allocate temp storage in shared memory
    __shared__ typename BlockReduce::TempStorage temp_storage;

    // Per-thread tile data
    T data[ITEMS_PER_THREAD];

    // Load first tile of data
    int block_offset = 0;
    LoadDirectBlocked(threadIdx.x, d_in + block_offset, data);
    block_offset += TILE_SIZE;

    // Start cycle timer
    clock_t start = clock();

    // Cooperative reduce first tile
    T block_aggregate = DeviceTest<BlockReduce>(temp_storage, data, reduction_op);

    // Stop cycle timer
#if CUB_PTX_VERSION == 100
    // Bug: recording stop clock causes mis-write of running prefix value
    clock_t stop = 0;
#else
    clock_t stop = clock();
#endif // CUB_PTX_VERSION == 100
    clock_t elapsed = (start > stop) ? start - stop : stop - start;

    // Loop over input tiles
    while (block_offset < TILE_SIZE * tiles)
    {
        // TestBarrier between threadblock reductions
        __syncthreads();

        // Load tile of data
        LoadDirectBlocked(threadIdx.x, d_in + block_offset, data);
        block_offset += TILE_SIZE;

        // Start cycle timer
        clock_t start = clock();

        // Cooperatively reduce the tile's aggregate
        T tile_aggregate = DeviceTest<BlockReduce>(temp_storage, data, reduction_op);

        // Stop cycle timer
    #if CUB_PTX_VERSION == 100
        // Bug: recording stop clock causes mis-write of running prefix value
        clock_t stop = 0;
    #else
        clock_t stop = clock();
    #endif // CUB_PTX_VERSION == 100
        elapsed += (start > stop) ? start - stop : stop - start;

        // Reduce threadblock aggregate
        block_aggregate = reduction_op(block_aggregate, tile_aggregate);
    }

    // Store data
    if (threadIdx.x == 0)
    {
        d_out[0] = block_aggregate;
        *d_elapsed = elapsed;
    }
}



/**
 * Test partial-tile reduction kernel (where num_items < BLOCK_THREADS)
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
__launch_bounds__ (BLOCK_THREADS, 1)
__global__ void PartialTileReduceKernel(
    T                       *d_in,
    T                       *d_out,
    int                     num_items,
    ReductionOp             reduction_op,
    clock_t                 *d_elapsed)
{
    // Cooperative threadblock reduction utility type (returns aggregate only in thread-0)
    typedef BlockReduce<T, BLOCK_THREADS, ALGORITHM> BlockReduce;

    // Allocate temp storage in shared memory
    __shared__ typename BlockReduce::TempStorage temp_storage;

    // Per-thread tile data
    T partial;

    // Load partial tile data
    if (threadIdx.x < num_items)
    {
        partial = d_in[threadIdx.x];
    }

    // Start cycle timer
    clock_t start = clock();

    // Cooperatively reduce the tile's aggregate
    T tile_aggregate = DeviceTest<BlockReduce>(temp_storage, partial, reduction_op, num_items);

    // Stop cycle timer
#if CUB_PTX_VERSION == 100
    // Bug: recording stop clock causes mis-write of running prefix value
    clock_t stop = 0;
#else
    clock_t stop = clock();
#endif // CUB_PTX_VERSION == 100
    clock_t elapsed = (start > stop) ? start - stop : stop - start;

    // Store data
    if (threadIdx.x == 0)
    {
        d_out[0] = tile_aggregate;
        *d_elapsed = elapsed;
    }
}


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize problem (and solution)
 */
template <
    typename    T,
    typename    ReductionOp>
void Initialize(
    GenMode     gen_mode,
    T           *h_in,
    T           h_reference[1],
    ReductionOp reduction_op,
    int         num_items)
{
    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
        if (i == 0)
            h_reference[0] = h_in[0];
        else
            h_reference[0] = reduction_op(h_reference[0], h_in[i]);
    }
}


//---------------------------------------------------------------------
// Full tile test generation
//---------------------------------------------------------------------


/**
 * Test full-tile reduction
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    typename                T,
    typename                ReductionOp>
void TestFullTile(
    GenMode                 gen_mode,
    int                     tiles,
    ReductionOp             reduction_op,
    char                    *type_string)
{
    const int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

    int num_items = TILE_SIZE * tiles;

    // Allocate host arrays
    T *h_in = new T[num_items];
    T h_reference[1];

    // Initialize problem
    Initialize(gen_mode, h_in, h_reference, reduction_op, num_items);

    // Initialize/clear device arrays
    T       *d_in = NULL;
    T       *d_out = NULL;
    clock_t *d_elapsed = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_elapsed, sizeof(unsigned long long)));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * 1));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * 1));

    // Test multi-tile (unguarded)
    printf("TestFullTile %s, gen-mode %d, num_items(%d), BLOCK_THREADS(%d), ITEMS_PER_THREAD(%d), tiles(%d), %s (%d bytes) elements:\n",
        (ALGORITHM == BLOCK_REDUCE_RAKING) ? "BLOCK_REDUCE_RAKING" : (ALGORITHM == BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY) ? "BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY" : "BLOCK_REDUCE_WARP_REDUCTIONS",
        gen_mode,
        num_items,
        BLOCK_THREADS,
        ITEMS_PER_THREAD,
        tiles,
        type_string,
        (int) sizeof(T));
    fflush(stdout);

    FullTileReduceKernel<ALGORITHM, BLOCK_THREADS, ITEMS_PER_THREAD><<<1, BLOCK_THREADS>>>(
        d_in,
        d_out,
        reduction_op,
        tiles,
        d_elapsed);

    CubDebugExit(hipDeviceSynchronize());

    // Copy out and display results
    printf("\tReduction results: ");
    int compare = CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    printf("\tElapsed clocks: ");
    DisplayDeviceResults(d_elapsed, 1);

    // Cleanup
    if (h_in) delete[] h_in;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_elapsed) CubDebugExit(g_allocator.DeviceFree(d_elapsed));
}

/**
 * Run battery of tests for different thread items
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
void TestFullTile(
    GenMode                 gen_mode,
    int                     tiles,
    ReductionOp             reduction_op,
    char                    *type_string)
{
    TestFullTile<ALGORITHM, BLOCK_THREADS, 1, T>(gen_mode, tiles, reduction_op, type_string);
    TestFullTile<ALGORITHM, BLOCK_THREADS, 4, T>(gen_mode, tiles, reduction_op, type_string);
}


/**
 * Run battery of full-tile tests for different numbers of tiles
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
void TestFullTile(
    GenMode                 gen_mode,
    ReductionOp             reduction_op,
    char                    *type_string)
{
    for (int tiles = 1; tiles < 3; tiles++)
    {
        TestFullTile<ALGORITHM, BLOCK_THREADS, T>(gen_mode, tiles, reduction_op, type_string);
    }
}


//---------------------------------------------------------------------
// Partial-tile test generation
//---------------------------------------------------------------------

/**
 * Test partial-tile reduction
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
void TestPartialTile(
    GenMode                 gen_mode,
    int                     num_items,
    ReductionOp             reduction_op,
    char                    *type_string)
{
    const int TILE_SIZE = BLOCK_THREADS;

    // Allocate host arrays
    T *h_in = new T[num_items];
    T h_reference[1];

    // Initialize problem
    Initialize(gen_mode, h_in, h_reference, reduction_op, num_items);

    // Initialize/clear device arrays
    T       *d_in = NULL;
    T       *d_out = NULL;
    clock_t *d_elapsed = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_elapsed, sizeof(unsigned long long)));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * TILE_SIZE));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * 1));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * 1));

    printf("TestPartialTile %s, gen-mode %d, num_items(%d), BLOCK_THREADS(%d), %s (%d bytes) elements:\n",
        (ALGORITHM == BLOCK_REDUCE_RAKING) ? "BLOCK_REDUCE_RAKING" : (ALGORITHM == BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY) ? "BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY" : "BLOCK_REDUCE_WARP_REDUCTIONS",
        gen_mode,
        num_items,
        BLOCK_THREADS,
        type_string,
        (int) sizeof(T));
    fflush(stdout);

    PartialTileReduceKernel<ALGORITHM, BLOCK_THREADS><<<1, BLOCK_THREADS>>>(
        d_in,
        d_out,
        num_items,
        reduction_op,
        d_elapsed);

    CubDebugExit(hipDeviceSynchronize());

    // Copy out and display results
    printf("\tReduction results: ");
    int compare = CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    printf("\tElapsed clocks: ");
    DisplayDeviceResults(d_elapsed, 1);

    // Cleanup
    if (h_in) delete[] h_in;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_elapsed) CubDebugExit(g_allocator.DeviceFree(d_elapsed));
}


/**
 *  Run battery of partial-tile tests for different numbers of effective threads
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
void TestPartialTile(
    GenMode                 gen_mode,
    ReductionOp             reduction_op,
    char                    *type_string)
{
    for (
        int num_items = 1;
        num_items < BLOCK_THREADS;
        num_items += CUB_MAX(1, BLOCK_THREADS / 5))
    {
        TestPartialTile<ALGORITHM, BLOCK_THREADS, T>(gen_mode, num_items, reduction_op, type_string);
    }
}



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Run battery of full-tile tests for different gen modes
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
void Test(
    ReductionOp             reduction_op,
    char                    *type_string)
{
    TestFullTile<ALGORITHM, BLOCK_THREADS, T>(UNIFORM, reduction_op, type_string);
    TestPartialTile<ALGORITHM, BLOCK_THREADS, T>(UNIFORM, reduction_op, type_string);

    TestFullTile<ALGORITHM, BLOCK_THREADS, T>(INTEGER_SEED, reduction_op, type_string);
    TestPartialTile<ALGORITHM, BLOCK_THREADS, T>(INTEGER_SEED, reduction_op, type_string);

    if (Traits<T>::CATEGORY != FLOATING_POINT)
    {
        // Don't test randomly-generated floats b/c of stability
        TestFullTile<ALGORITHM, BLOCK_THREADS, T>(RANDOM, reduction_op, type_string);
        TestPartialTile<ALGORITHM, BLOCK_THREADS, T>(RANDOM, reduction_op, type_string);
    }
}


/**
 * Run battery of tests for different block-reduction algorithmic variants
 */
template <
    int             BLOCK_THREADS,
    typename        T,
    typename        ReductionOp>
void Test(
    ReductionOp     reduction_op,
    char            *type_string)
{
    Test<BLOCK_REDUCE_RAKING, BLOCK_THREADS, T>(reduction_op, type_string);
    Test<BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY, BLOCK_THREADS, T>(reduction_op, type_string);
    Test<BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_THREADS, T>(reduction_op, type_string);
}


/**
 * Run battery of tests for different block sizes
 */
template <
    typename        T,
    typename        ReductionOp>
void Test(
    ReductionOp     reduction_op,
    char            *type_string)
{
    Test<7, T>(reduction_op, type_string);
    Test<32, T>(reduction_op, type_string);
    Test<63, T>(reduction_op, type_string);
    Test<97, T>(reduction_op, type_string);
    Test<128, T>(reduction_op, type_string);
    Test<238, T>(reduction_op, type_string);
}


/**
 * Run battery of tests for different block sizes
 */
template <typename T>
void Test(char* type_string)
{
    Test<T>(Sum(), type_string);
    Test<T>(Max(), type_string);
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("repeat", g_repeat);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--repeat=<repetitions of entire test suite>]"
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Get ptx version
    int ptx_version;
    CubDebugExit(PtxVersion(ptx_version));

#ifdef QUICK_TEST

    // Compile/run quick tests
    typedef int T;

    printf("\n full tile ------------------------\n\n");

    TestFullTile<BLOCK_REDUCE_RAKING,                   128, 4, int>(UNIFORM, 1, Sum(), CUB_TYPE_STRING(int));
    TestFullTile<BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,  128, 4, int>(UNIFORM, 1, Sum(), CUB_TYPE_STRING(int));
    TestFullTile<BLOCK_REDUCE_WARP_REDUCTIONS,          128, 4, int>(UNIFORM, 1, Sum(), CUB_TYPE_STRING(int));

    TestFullTile<BLOCK_REDUCE_RAKING,                   128, 1, int>(UNIFORM, 1, Sum(), CUB_TYPE_STRING(int));
    TestFullTile<BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,  128, 1, int>(UNIFORM, 1, Sum(), CUB_TYPE_STRING(int));
    TestFullTile<BLOCK_REDUCE_WARP_REDUCTIONS,          128, 1, int>(UNIFORM, 1, Sum(), CUB_TYPE_STRING(int));

    printf("\n partial tile ------------------------\n\n");

    TestPartialTile<BLOCK_REDUCE_RAKING,                   128, int>(UNIFORM, 1, Sum(), CUB_TYPE_STRING(int));
    TestPartialTile<BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,  128, int>(UNIFORM, 1, Sum(), CUB_TYPE_STRING(int));
    TestPartialTile<BLOCK_REDUCE_WARP_REDUCTIONS,          128, int>(UNIFORM, 1, Sum(), CUB_TYPE_STRING(int));

#else

    // Compile/run thorough tests
    for (int i = 0; i <= g_repeat; ++i)
    {
        // primitives
        Test<char>(CUB_TYPE_STRING(char));
        Test<short>(CUB_TYPE_STRING(short));
        Test<int>(CUB_TYPE_STRING(int));
        Test<long long>(CUB_TYPE_STRING(long long));
        if (ptx_version > 100)                          // Don't check doubles on PTX100 because they're down-converted
            Test<double>(CUB_TYPE_STRING(double));

        // vector types
        Test<char2>(CUB_TYPE_STRING(char2));
        Test<short2>(CUB_TYPE_STRING(short2));
        Test<int2>(CUB_TYPE_STRING(int2));
        Test<longlong2>(CUB_TYPE_STRING(longlong2));

        Test<char4>(CUB_TYPE_STRING(char4));
        Test<short4>(CUB_TYPE_STRING(short4));
        Test<int4>(CUB_TYPE_STRING(int4));
        Test<longlong4>(CUB_TYPE_STRING(longlong4));

        // Complex types
        Test<TestFoo>(CUB_TYPE_STRING(TestFoo));
        Test<TestBar>(CUB_TYPE_STRING(TestBar));
    }

#endif

    return 0;
}



